#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This code implements the interleaved Pair approaches to
 * parallel reduction in CUDA. For this example, the sum operation is used.
 */


// implemented q dependend kernel function

__global__ void reduceUnrolling (double *g_idata, double *g_odata, unsigned int n, unsigned int q) //added int q
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * q + threadIdx.x; // q adapted idx

    // unroll analogous q
    if (idx + blockDim.x*(q-1) < n)
    {
      for (int i=1; i<q; i++)
      {
	g_idata[idx] += g_idata[idx + blockDim.x*i];
      }
    }
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (tid < stride)
        {
            g_idata[idx] += g_idata[idx + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = g_idata[idx];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;

    // initialization
    int size = 1 << 24; // total number of elements to reduce
    printf("    with array size %d  ", size);

    // execution configuration
    int blocksize = 1024;   // initial block size
    int q = 2048;
    if(argc > 2)
    {
        blocksize = atoi(argv[1]);   // block size from command line argument
        q = atoi(argv[2]);   // block size from command line argument
    }

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(double);
    double *h_idata = (double *) malloc(bytes);
    double *h_odata = (double *) malloc(grid.x * sizeof(double));
    double *tmp     = (double *) malloc(bytes);

    // initialize the array
    int sign=1;
    for (int i = 0; i < size; i++)
    {
        // mask off high 2 bytes to force max number to 255
        h_idata[i] = sign*((double)( rand() & 0xFF ));
        sign*=-1;
    }

    memcpy (tmp, h_idata, bytes);

    double iStart, iElaps;
    double gpu_sum = 0.0;

    // allocate device memory
    double *d_idata = NULL;
    double *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(double)));

      // kernel: reduceUnrolling optimized with q
    if (grid.x>1)
    {  
 
       dim3 gridq ((grid.x + 1)/q,1); // change grid dim due to q
       CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
       CHECK(hipDeviceSynchronize());
       iStart = seconds();
       reduceUnrolling<<<gridq.x, block>>>(d_idata, d_odata, size,q); // call optimized kernel function w. q
       CHECK(hipDeviceSynchronize());
       iElaps = seconds() - iStart;
       CHECK(hipGetLastError());
       CHECK(hipMemcpy(h_odata, d_odata, gridq.x * sizeof(double),
                        hipMemcpyDeviceToHost));
       gpu_sum = 0;

       for (int i = 0; i < gridq.x; i++) gpu_sum += h_odata[i];

       printf("gpu Unrolling optimized w. q = %d  elapsed %f sec gpu_sum: %d <<<grid %d block "
              "%d>>>\n", q, iElaps, gpu_sum, gridq.x, block.x);
      }
    // free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    // reset device
    CHECK(hipDeviceReset());

    // check the results
    bResult = (gpu_sum == cpu_sum);

    if(!bResult) printf("Test failed!\n");

    return EXIT_SUCCESS;
}
