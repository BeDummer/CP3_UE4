#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * This code implements the interleaved Pair approaches to
 * parallel reduction in CUDA. For this example, the sum operation is used.
 */

// Recursive Implementation of Interleaved Pair Approach
int recursiveReduce(int *data, int const size)
{
    // terminate check
    if (size == 1) return data[0];

    // renew the stride
    int const stride = size / 2;

    // in-place reduction
    for (int i = 0; i < stride; i++)
    {
        data[i] += data[i + stride];
    }

    // call recursively
    return recursiveReduce(data, stride);
}

// Kernel: Interleaved Pair Implementation
__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if(idx >= n) return;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (tid < stride)
        {
            g_idata[idx] += g_idata[idx + stride];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = g_idata[idx];
}

__global__ void reduceUnrolling (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // unroll 2
    if (idx + blockDim.x < n)
    {
        g_idata[idx] += g_idata[idx + blockDim.x];
    }
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (tid < stride)
        {
            g_idata[idx] += g_idata[idx + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = g_idata[idx];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting reduction at ", argv[0]);
    printf("device %d: %s ", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    bool bResult = false;

    // initialization
    int size = 1 << 24; // total number of elements to reduce
    printf("    with array size %d  ", size);

    // execution configuration
    int blocksize = 512;   // initial block size

    if(argc > 1)
    {
        blocksize = atoi(argv[1]);   // block size from command line argument
    }

    dim3 block (blocksize, 1);
    dim3 grid  ((size + block.x - 1) / block.x, 1);
    printf("grid %d block %d\n", grid.x, block.x);

    // allocate host memory
    size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp     = (int *) malloc(bytes);

    // initialize the array
    int sign=1;
    for (int i = 0; i < size; i++)
    {
        // mask off high 2 bytes to force max number to 255
        h_idata[i] = sign*((int)( rand() & 0xFF ));
        sign*=-1;
    }

    memcpy (tmp, h_idata, bytes);

    double iStart, iElaps;
    int gpu_sum = 0;

    // allocate device memory
    int *d_idata = NULL;
    int *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(int)));

    // cpu reduction
    iStart = seconds();
    int cpu_sum = recursiveReduce (tmp, size);
    iElaps = seconds() - iStart;
    printf("cpu reduce      elapsed %f sec cpu_sum: %d\n", iElaps, cpu_sum);

    // kernel: reduceInterleaved
    CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    reduceInterleaved<<<grid, block>>>(d_idata, d_odata, size);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    CHECK(hipGetLastError());
    CHECK(hipMemcpy(h_odata, d_odata, grid.x * sizeof(int),
                     hipMemcpyDeviceToHost));
    gpu_sum = 0;

    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    printf("gpu Interleaved elapsed %f sec gpu_sum: %d <<<grid %d block "
           "%d>>>\n", iElaps, gpu_sum, grid.x, block.x);

    // kernel: reduceUnrolling
    if (grid.x>1)
    {
       dim3 grid2 ((grid.x + 1)/2,1);
       CHECK(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
       CHECK(hipDeviceSynchronize());
       iStart = seconds();
       reduceUnrolling<<<grid2.x, block>>>(d_idata, d_odata, size);
       CHECK(hipDeviceSynchronize());
       iElaps = seconds() - iStart;
       CHECK(hipGetLastError());
       CHECK(hipMemcpy(h_odata, d_odata, grid2.x * sizeof(int),
                        hipMemcpyDeviceToHost));
       gpu_sum = 0;

       for (int i = 0; i < grid2.x; i++) gpu_sum += h_odata[i];

       printf("gpu Unrolling  elapsed %f sec gpu_sum: %d <<<grid %d block "
              "%d>>>\n", iElaps, gpu_sum, grid2.x, block.x);
    }

    // free host memory
    free(h_idata);
    free(h_odata);

    // free device memory
    CHECK(hipFree(d_idata));
    CHECK(hipFree(d_odata));

    // reset device
    CHECK(hipDeviceReset());

    // check the results
    bResult = (gpu_sum == cpu_sum);

    if(!bResult) printf("Test failed!\n");

    return EXIT_SUCCESS;
}
